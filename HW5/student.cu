#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/

// 9ms original 


#include "utils.h"
#include <stdio.h>

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
  size_t id = blockIdx.x * blockDim.x + threadIdx.x;
  if(id >= numVals)
    return;
  atomicAdd(histo + vals[id],1);
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                      unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel

  //if you want to use/launch more than one kernel,
  //feel free
  size_t threadsPerBlock = 256;
  size_t blocks = ceil(float(numElems)/256);
  
  printf("Number of bins: %u\n", numBins);
  
  yourHisto<<<blocks, threadsPerBlock>>>(d_vals, d_histo, numElems);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
}
